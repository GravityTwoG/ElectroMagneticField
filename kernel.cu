#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <GL/glew.h>
#include <GL/freeglut.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>


const GLint WINDOW_WIDTH = 820;
const GLint WINDOW_HEIGHT = 640;
const int4 magneticField = { WINDOW_WIDTH / 3, 0, WINDOW_WIDTH, WINDOW_HEIGHT };

__device__ const float starB = 1e-9; // Tsl
__device__ const float2 starE = { 0, 8e-6 }; // V/m
__device__ const float C = 3e8;      // m/s
__device__ const float lambda = 3;// m
__device__ const int4 d_magneticField = { 
	WINDOW_WIDTH/3, // x-start
	-WINDOW_HEIGHT*10,              // y-start
	WINDOW_WIDTH*10,   // x-end
	WINDOW_HEIGHT*10   // y-end
};

const float TIME_SCALE = 0.5;
const float starV = 5e4; // m/s
const float V_MAX = starV / C;
const float V_MIN = 0.3 * V_MAX;

const float MAX_CHARGE = 1.6e-19;
const float MIN_CHARGE = 0.3 * MAX_CHARGE;
const int MAX_CHARGE_COUNT = 120;
__constant__ const float K = 2e21;
__constant__ const float MIN_DISTANCE = 1.0f; // not to divide by zero

int chargeCount = 0;
__device__ int dev_chargeCount;

struct Particle {
	float x;
	float y;
	float vx;
	float vy;
	float charge;
	float mass;
	bool isPhysical;
};

Particle charges[MAX_CHARGE_COUNT]; 
Particle* dev_charges;

/* OpenGL interoperability */
dim3 blocks, threads;
GLuint vbo;
hipGraphicsResource* cuda_vbo_resource;

static void cudaCheckError(hipError_t err, const char* file, int line);
#define HANDLE_ERROR( err ) (cudaCheckError( err, __FILE__, __LINE__ ))

void createVBO(GLuint* vbo, hipGraphicsResource** vbo_res, unsigned int vbo_res_flags);
void deleteVBO(GLuint* vbo, hipGraphicsResource* vbo_res);


__device__ bool isInMagneticField(float x, float y) {
	if (x < d_magneticField.x) return false;
	if (x > d_magneticField.z) return false;
	if (y < d_magneticField.y) return false;
	if (y > d_magneticField.w) return false;

	return true;
}

__device__ inline float4 dF(const Particle& p) {
	if (isInMagneticField(p.x, p.y)) {
		float k = p.charge * lambda / (p.mass * C);
		float B = starB * k;
		float2 E = { 
			starE.x * k / C, 
			starE.y * k / C
		};

		return {
			p.vx,
			p.vy,
			E.x + B * p.vy,
			E.y - B * p.vx
		};
	}

	return {
		p.vx,
		p.vy,
		0,
		0
	};
}

// apply Columbus Law
__global__ void dev_applyMagneticField(uchar4* screen, Particle* dev_charges, float dt) {
	int charge_i = blockIdx.x * blockDim.x + threadIdx.x;
	if (charge_i >= dev_chargeCount) return;

	Particle& particle = dev_charges[charge_i];
	if (!particle.isPhysical) return;

	//float4 fi = dF(particle);
	//particle.x += dt * fi.x; // x + dx
	//particle.y += dt * fi.y; // y + dy
	//if (isInMagneticField(particle.x, particle.y)) {
	//	particle.vx += dt * fi.z;
	//	particle.vy += dt * fi.w;
	//}

	Particle p2 = particle;
	float4 d1 = dF(p2);

	p2.x = particle.x + dt * d1.x / 2;
	p2.y = particle.y + dt * d1.y / 2;
	p2.vx = particle.vx + dt * d1.z / 2;
	p2.vy = particle.vy + dt * d1.w / 2;
	float4 d2 = dF(p2);

	p2.x = particle.x + dt * d2.x / 2;
	p2.y = particle.y + dt * d2.y / 2;
	p2.vx = particle.vx + dt * d2.z / 2;
	p2.vy = particle.vy + dt * d2.w / 2;
	float4 d3 = dF(p2);
	
	p2.x = particle.x + dt * d3.x;
	p2.y = particle.y + dt * d3.y;
	p2.vx = particle.vx + dt * d3.z;
	p2.vy = particle.vy + dt * d3.w;
	float4 d4 = dF(p2);

	particle.x += dt / 6 * (d1.x + 2*d2.x + 2*d3.x + d4.x); // x + dx
	particle.y += dt / 6 * (d1.y + 2*d2.y + 2*d3.y + d4.y); // y + dy
	particle.vx += dt / 6 * (d1.z + 2 * d2.z + 2 * d3.z + d4.z);
	particle.vy += dt / 6 * (d1.w + 2 * d2.w + 2 * d3.w + d4.w);

	if (particle.x >= 10 * WINDOW_WIDTH) particle.isPhysical = false;
	if (particle.x < -10 * WINDOW_WIDTH) particle.isPhysical = false;
	if (particle.y >= 10 * WINDOW_HEIGHT) particle.isPhysical = false;
	if (particle.y < -10 * WINDOW_HEIGHT) particle.isPhysical = false;

	if (
		particle.x < WINDOW_WIDTH && 
		particle.x >=  0 && 
		particle.y < WINDOW_HEIGHT && 
		particle.y >= 0
	) {
		uchar4& pixel = screen[(int)particle.x + (int)particle.y * WINDOW_WIDTH];
		pixel.y = 250;
	}
}

__global__ void dev_clearFrame(uchar4* screen) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= WINDOW_WIDTH || y >= WINDOW_HEIGHT) return;

	uchar4& pixel = screen[x + y * WINDOW_WIDTH];
	pixel.x = 0;
	pixel.y = 0;
	pixel.z = 0;
	pixel.w = 255;
}

// Compute electric field
__global__ void dev_renderFrame(uchar4* screen, Particle* dev_charges) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= WINDOW_WIDTH || y >= WINDOW_HEIGHT) return;

	float2 force = { 0, 0 };
	float E = 0;
	// iterate over all charges and compute resulted force vector
	for (char i = 0; i < dev_chargeCount; i++) {
		const Particle& particle = dev_charges[i];
		float2 t_force = {
			x - particle.x, // dx
			y - particle.y  // dy
		};

		float lengthSquared = t_force.x*t_force.x + t_force.y*t_force.y + MIN_DISTANCE;

		//e = q / (x^2 + y^2)^(3/2)
		float e = particle.charge / sqrtf(lengthSquared * lengthSquared * lengthSquared);
		E += e;
		t_force.x *= e;
		t_force.y *= e;

		force.x += t_force.x;
		force.y += t_force.y;
	}

	force.x *= K;
	force.y *= K;

	uchar4& pixel = screen[x + y * WINDOW_WIDTH];
	//pixel.x = pixel.z = 0;
	pixel.y = 0;
	pixel.w = 255;

	float l = sqrtf(force.x * force.x + force.y * force.y); // 
	//if (l < 70) return;

	float lScale = 2;
	int brightness = l * lScale;
	if (E > 0.0) {
		pixel.x = pixel.x > brightness ? pixel.x : brightness;
	} else {
		pixel.z = pixel.z > brightness ? pixel.z : brightness;
	}
}

float elapsedTime = 0.0f;

void idle(void) {
	uchar4* dev_screen;
	size_t size;

	HANDLE_ERROR(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
	HANDLE_ERROR(
		hipGraphicsResourceGetMappedPointer((void**)&dev_screen, &size, cuda_vbo_resource)
	);

	// Kernel Time measure
	hipEvent_t startEvent, stopEvent;
	HANDLE_ERROR(hipEventCreate(&startEvent));
	HANDLE_ERROR(hipEventCreate(&stopEvent));
	HANDLE_ERROR(hipEventRecord(startEvent, 0));

	//float elapsedTimeS = elapsedTime / 1000.0;
	float elapsedTimeS = 1 / 1000.0;
	float dtau = elapsedTimeS * C / lambda;
	dev_renderFrame<<<blocks, threads>>>(dev_screen, dev_charges);
	dev_applyMagneticField<<<1, MAX_CHARGE_COUNT>>>(
		dev_screen, dev_charges, 
		dtau * TIME_SCALE
	);
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));

	// Kernel Time measure
	HANDLE_ERROR(hipEventRecord(stopEvent, 0));
	HANDLE_ERROR(hipEventSynchronize(stopEvent));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent));

	char fps[256];
	sprintf(fps, "%3.2f ms per frame (FPS: %3.1f)", elapsedTime,
		1000 / elapsedTime);
	glutSetWindowTitle(fps);

	glutPostRedisplay();
}

void draw(void) {
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	
	glDrawPixels(WINDOW_WIDTH, WINDOW_HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	glColor4f(0.4f, 0.4f, 1.0f, 0.2f);
	glRecti(
		magneticField.x, magneticField.y,
		magneticField.z, magneticField.w
	);

	glutSwapBuffers();
}

void clearScreen() {
	for (int i = 0; i < chargeCount; i++) {
		charges[i].isPhysical = false;
	}
	HANDLE_ERROR(
		hipMemcpy(dev_charges, charges, chargeCount * sizeof(Particle), hipMemcpyHostToDevice)
	);
	chargeCount = 0;
	HANDLE_ERROR(
		hipMemcpyToSymbol(HIP_SYMBOL(dev_chargeCount), &chargeCount, sizeof(chargeCount))
	);
	uchar4* dev_screen;
	size_t size;

	HANDLE_ERROR(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
	HANDLE_ERROR(
		hipGraphicsResourceGetMappedPointer((void**)&dev_screen, &size, cuda_vbo_resource)
	);
	dev_clearFrame<<<blocks, threads>>>(dev_screen);
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
	glutPostRedisplay();
}

void addCharge(int x, int y) {
	if (chargeCount < MAX_CHARGE_COUNT) {
		chargeCount++;
	} else {
		// remove first charge
		for (int i = 0; i < MAX_CHARGE_COUNT - 1; ++i) {
			charges[i] = charges[i + 1];
		}
	}

	float scale = rand() / (float)RAND_MAX; /* [0, 1.0] */
	float newCharge = MIN_CHARGE + (float)scale * (MAX_CHARGE - MIN_CHARGE);      /* [min, max] */

	float scale2 = rand() / (float)RAND_MAX; /* [0, 1.0] */
	if (scale2 < 0.5) {
		newCharge = -newCharge;
	}
	
	float vScale = rand() / (float)RAND_MAX; /* [0, 1.0] */

	charges[chargeCount - 1].x = x;
	charges[chargeCount - 1].y = y;
	charges[chargeCount - 1].charge = newCharge;
	charges[chargeCount - 1].vx = V_MIN + vScale * (V_MAX - V_MIN);
	charges[chargeCount - 1].vy = 0.0f;
	charges[chargeCount - 1].mass = fabs(newCharge / 10e10);
	charges[chargeCount - 1].mass = 9e-31;
	charges[chargeCount - 1].isPhysical = true;

	printf("Charges %d\n", chargeCount);
}

void addCharges(int x, int y, int n) {
	HANDLE_ERROR(
		hipMemcpy(charges, dev_charges, chargeCount * sizeof(Particle), hipMemcpyDeviceToHost)
	);
	
	float disp = 40;
	for (int i = 0; i < MAX_CHARGE_COUNT && i < n; i++) {
		float dx = rand() / (float)RAND_MAX * disp;
		float dy = rand() / (float)RAND_MAX * disp;
		
		addCharge(x + dx - disp/2, y + dy - disp / 2);
	}

	HANDLE_ERROR(
		hipMemcpy(dev_charges, charges, chargeCount * sizeof(Particle), hipMemcpyHostToDevice)
	);
	HANDLE_ERROR(
		hipMemcpyToSymbol(HIP_SYMBOL(dev_chargeCount), &chargeCount, sizeof(chargeCount))
	);
}


void onMouseEvent(int button, int state, int x, int y) {
	if (button == GLUT_MIDDLE_BUTTON && state == GLUT_DOWN) {
		clearScreen();
		return;
	}
	
	if (button == GLUT_RIGHT_BUTTON && state == GLUT_UP) {
		addCharges(x, WINDOW_HEIGHT - y, 1);
		return;
	}

	if (button == GLUT_LEFT_BUTTON && state == GLUT_UP) {
		clearScreen();
		addCharges(x, WINDOW_HEIGHT - y, MAX_CHARGE_COUNT);
		return;
	}
}

void onKeyEvent(unsigned char key, int x, int y) {
	switch (key) {
	case 27:
		printf("Exit application\n");

		glutLeaveMainLoop();
		break;
	}
}

void onResize(int width, int height) {
	glutReshapeWindow(WINDOW_WIDTH, WINDOW_HEIGHT);
}

void initCuda(int deviceId) {
	hipDeviceProp_t properties;
	HANDLE_ERROR(hipGetDeviceProperties(&properties, deviceId));

	threads.x = 32;
	threads.y = properties.maxThreadsPerBlock / threads.x - 2;

	blocks.x = (WINDOW_WIDTH + threads.x - 1) / threads.x;
	blocks.y = (WINDOW_HEIGHT + threads.y - 1) / threads.y;

	hipMalloc(&dev_charges, sizeof(Particle) * MAX_CHARGE_COUNT);

	printf(
		"Debug: blocks(%d, %d), threads(%d, %d)\nCalculated Resolution: %d x %d\n",
		blocks.x, blocks.y, threads.x, threads.y, blocks.x * threads.x,
		blocks.y * threads.y
	);
}

void initGlut(int argc, char** argv) {
	// Initialize freeglut
	glutInit(&argc, argv);
	
	//glutInitDisplayMode(GLUT_RGBA);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	
	glutInitWindowSize(WINDOW_WIDTH, WINDOW_HEIGHT);
	int posX = glutGet(GLUT_SCREEN_WIDTH) / 2 - WINDOW_WIDTH / 2;
	int posY = glutGet(GLUT_SCREEN_HEIGHT) / 2 - WINDOW_HEIGHT / 2;
	glutInitWindowPosition(posX, posY);
	glutCreateWindow("Lab-4");

	glutIdleFunc(idle);
	glutDisplayFunc(draw);
	glutKeyboardFunc(onKeyEvent);
	glutMouseFunc(onMouseEvent);
	glutReshapeFunc(onResize);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0, (GLdouble)WINDOW_WIDTH, 0.0, (GLdouble)WINDOW_HEIGHT);

	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glEnable(GL_BLEND);

	glewInit();
}

int main(int argc, char** argv) {
	srand(time(0));
	initCuda(0);
	initGlut(argc, argv);

	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	glutMainLoop();

	deleteVBO(&vbo, cuda_vbo_resource);

	return 0;
}

static void cudaCheckError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

void createVBO(
	GLuint* vbo,
	struct hipGraphicsResource** vbo_res,
	unsigned int vbo_res_flags
) {
	unsigned int size = WINDOW_WIDTH * WINDOW_HEIGHT * sizeof(uchar4);

	glGenBuffers(1, vbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, *vbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, size, NULL, GL_DYNAMIC_DRAW);

	HANDLE_ERROR(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));
}

void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res) {
	HANDLE_ERROR(hipGraphicsUnregisterResource(cuda_vbo_resource));

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}