#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <GL/glew.h>
#include <GL/freeglut.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>


const GLint WINDOW_WIDTH = 820;
const GLint WINDOW_HEIGHT = 640;
const int4 magneticField = { WINDOW_WIDTH / 3, 0, WINDOW_WIDTH, WINDOW_HEIGHT };

__device__ const float starB = -5e-10; // Tsl
__device__ const float C = 3e8;      // m/s
__device__ const float lambda = 1;// m
__device__ const int4 d_magneticField = { 
	WINDOW_WIDTH/3, // x-start
	-WINDOW_HEIGHT*10,              // y-start
	WINDOW_WIDTH*10,   // x-end
	WINDOW_HEIGHT*10   // y-end
};

const float starV = 5e4; // m/s
const float V = starV / C;

/* charge constants */
__constant__ const float K = 1e20;
__constant__ float MIN_DISTANCE = 1.0f; // not to divide by zero

const float MIN_CHARGE = 0.2e-19;
const float MAX_CHARGE = 1.6e-19;
const char MAX_CHARGE_COUNT = 30;

char chargeCount = 0;
__constant__ char dev_chargeCount;

struct Particle {
	float x;
	float y;
	float vx;
	float vy;
	float charge;
	float mass;
	bool isPhysical;
};

Particle charges[MAX_CHARGE_COUNT]; 
Particle* dev_charges;

/* OpenGL interoperability */
dim3 blocks, threads;
GLuint vbo;
hipGraphicsResource* cuda_vbo_resource;

/* charge selection */
const int DETECT_CHARGE_RANGE = 10;
int selectedChargeIndex = -1;
bool isDragging = false;

static void cudaCheckError(hipError_t err, const char* file, int line);
#define HANDLE_ERROR( err ) (cudaCheckError( err, __FILE__, __LINE__ ))

void createVBO(GLuint* vbo, hipGraphicsResource** vbo_res, unsigned int vbo_res_flags);
void deleteVBO(GLuint* vbo, hipGraphicsResource* vbo_res);

void onKeyEvent(unsigned char key, int x, int y) {
	switch (key) {
	case 27:
		printf("Exit application\n");

		glutLeaveMainLoop();
		break;
	}
}

__device__ float length(const float2& q) {
	return sqrtf(q.x * q.x + q.y * q.y);
}

__device__ float length2(const float2& q) {
	return q.x * q.x + q.y * q.y;
}

__device__ bool isInMagneticField(float x, float y) {
	if (x < d_magneticField.x) return false;
	if (x > d_magneticField.z) return false;
	if (y < d_magneticField.y) return false;
	if (y > d_magneticField.w) return false;

	return true;
}

// apply Columbus Law
__global__ void dev_applyMagneticField(uchar4* screen, Particle* dev_charges, float dt) {
	int charge_i = blockIdx.x * blockDim.x + threadIdx.x;
	if (charge_i >= MAX_CHARGE_COUNT) return;

	Particle& particle = dev_charges[charge_i];
	if (!particle.isPhysical) return;

	float2 v = { particle.vx, particle.vy };

	particle.x += dt * v.x;
	particle.y += dt * v.y;
	if (isInMagneticField(particle.x, particle.y)) {
		float B = starB * particle.charge * lambda / (particle.mass * C);
		particle.vx +=  dt * B * v.y;
		particle.vy += -dt * B * v.x;
	}

	if (particle.x >= 10 * WINDOW_WIDTH) particle.isPhysical = false;
	if (particle.x < -10 * WINDOW_WIDTH) particle.isPhysical = false;
	if (particle.y >= 10 * WINDOW_HEIGHT) particle.isPhysical = false;
	if (particle.y < -10 * WINDOW_HEIGHT) particle.isPhysical = false;

	if (
		particle.x < WINDOW_WIDTH && 
		particle.x >=  0 && 
		particle.y < WINDOW_HEIGHT && 
		particle.y >= 0
	) {
		uchar4& pixel = screen[(int)particle.x + (int)particle.y * WINDOW_WIDTH];
		//pixel.y = 255;
	}
}

// apply Columbus Law
__global__ void dev_applyElectricField(uchar4* screen, Particle* dev_charges, float dt) {
	int charge_i = blockIdx.x * blockDim.x + threadIdx.x;
	if (charge_i >= MAX_CHARGE_COUNT) return;

	Particle& currentParticle = dev_charges[charge_i];
	if (!currentParticle.isPhysical) return;

	float2 force = { 0.0f, 0.0f };
	// iterate over all paricles and compute resulted force vector
	for (char i = 0; i < dev_chargeCount; i++) {
		const Particle& particle = dev_charges[i];
		float2 t_force = {
			currentParticle.x - particle.x,
			currentParticle.y - particle.y
		};

		float lengthSquared = length2(t_force) + MIN_DISTANCE;
		float e = particle.charge / sqrt(lengthSquared * lengthSquared * lengthSquared);
		t_force.x *= e;
		t_force.y *= e;

		force.x += t_force.x;
		force.y += t_force.y;
	}

	const float localK = K * currentParticle.charge / currentParticle.mass / 1000.0;
	force.x = force.x * localK * currentParticle.vx;
	force.y = force.y * localK * currentParticle.vx;

	__syncthreads();

	currentParticle.vx += dt * currentParticle.x;
	currentParticle.vy += dt * currentParticle.y;
	currentParticle.x += dt * force.x;
	currentParticle.y += dt * force.y;

	if (currentParticle.x >= WINDOW_WIDTH) currentParticle.x = WINDOW_WIDTH - 1;
	if (currentParticle.x < 0) currentParticle.x = 0;
	if (currentParticle.y >= WINDOW_HEIGHT) currentParticle.y = WINDOW_HEIGHT - 1;
	if (currentParticle.y < 0) currentParticle.y = 0;
}

__global__ void dev_clearFrame(uchar4* screen) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= WINDOW_WIDTH || y >= WINDOW_HEIGHT) return;

	uchar4& pixel = screen[x + y * WINDOW_WIDTH];
	pixel.x = 0;
	pixel.y = 0;
	pixel.z = 0;
	pixel.w = 255;
}

// Compute electric field
__global__ void dev_renderFrame(uchar4* screen, Particle* dev_charges) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= WINDOW_WIDTH || y >= WINDOW_HEIGHT) return;

	float2 force;
	force.x = force.y = 0.0f;

	float E = 0;
	float2 t_force;
	// iterate over all charges and compute resulted force vector
	for (char i = 0; i < dev_chargeCount; i++) {
		const Particle& particle = dev_charges[i];
		t_force.x = x - particle.x; // dx
		t_force.y = y - particle.y; // dy

		// x^2 + y^2
		float lengthSquared = length2(t_force) + MIN_DISTANCE;

		//e = q / (x^2 + y^2)^(3/2)
		float e = particle.charge / sqrtf(lengthSquared * lengthSquared * lengthSquared);
		E += e;
		t_force.x *= e;
		t_force.y *= e;

		force.x += t_force.x;
		force.y += t_force.y;
	}

	force.x *= K;
	force.y *= K;

	// set color on current pixel
	uchar4& pixel = screen[x + y * WINDOW_WIDTH];
	//pixel.x = pixel.y = pixel.z = 0;
	pixel.w = 255;

	float l = length(force); // 
	if (l < 0.2) return;

	float maxL = 1.0;
	if (E > 0.0) {
		pixel.x = l > maxL ? 255 : l/maxL * 255;
	} else {
		pixel.z = l > maxL ? 255 : l/maxL * 255 ;
	}
}

float elapsedTime = 0.0f;

void idle(void) {
	uchar4* dev_screen;
	size_t size;

	HANDLE_ERROR(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
	HANDLE_ERROR(
		hipGraphicsResourceGetMappedPointer((void**)&dev_screen, &size, cuda_vbo_resource)
	);

	// Kernel Time measure
	hipEvent_t startEvent, stopEvent;
	HANDLE_ERROR(hipEventCreate(&startEvent));
	HANDLE_ERROR(hipEventCreate(&stopEvent));
	HANDLE_ERROR(hipEventRecord(startEvent, 0));

	// Compute Image
	dev_applyMagneticField<<<1, MAX_CHARGE_COUNT>>>(
		dev_screen, dev_charges, 
		elapsedTime / 1000.0 * C / lambda
	);
	//dev_applyElectricField<<<1, MAX_CHARGE_COUNT>>>(dev_screen, dev_charges, elapsedTime / 1.0);
	dev_renderFrame<<<blocks, threads>>>(dev_screen, dev_charges);
	HANDLE_ERROR(hipDeviceSynchronize());


	HANDLE_ERROR(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));

	// Kernel Time measure
	HANDLE_ERROR(hipEventRecord(stopEvent, 0));
	HANDLE_ERROR(hipEventSynchronize(stopEvent));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent));

	char fps[256];
	sprintf(fps, "%3.2f ms per frame (FPS: %3.1f)", elapsedTime,
		1000 / elapsedTime);
	glutSetWindowTitle(fps);

	glutPostRedisplay();
}

void draw(void) {
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	
	// Draw electric field
	glDrawPixels(WINDOW_WIDTH, WINDOW_HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	glColor4f(0.4f, 0.4f, 1.0f, 0.2f);
	glRecti(
		magneticField.x, magneticField.y,
		magneticField.z, magneticField.w
	);

	// draw selected point
	glPointSize(3.0f);
	glColor3f(0.0f, 1.0f, 1.0f);
	glBegin(GL_POINTS);
		glVertex2i(
			charges[selectedChargeIndex].x, 
			charges[selectedChargeIndex].y
		);
	glEnd();

	glutSwapBuffers();
}

void addCharge(int x, int y) {
	HANDLE_ERROR(
		hipMemcpy(charges, dev_charges, chargeCount * sizeof(Particle), hipMemcpyDeviceToHost)
	);
	
	if (chargeCount < MAX_CHARGE_COUNT) {
		chargeCount++;
	} else {
		// remove first charge
		for (int i = 0; i < MAX_CHARGE_COUNT - 1; ++i) {
			charges[i] = charges[i + 1];
		}
	}

	float scale = rand() / (float)RAND_MAX; /* [0, 1.0] */
	float newCharge = MIN_CHARGE + (float)scale * (MAX_CHARGE - MIN_CHARGE);      /* [min, max] */

	if (scale <= 0.5) {
		newCharge = -newCharge;
	}
	
	float vScale = rand() / (float)RAND_MAX; /* [0, 1.0] */

	charges[chargeCount - 1].x = x;
	charges[chargeCount - 1].y = y;
	charges[chargeCount - 1].charge = newCharge;
	charges[chargeCount - 1].vx = V * vScale;
	charges[chargeCount - 1].vy = 0.0f;
	charges[chargeCount - 1].mass = fabs(newCharge / 10e10);
	charges[chargeCount - 1].isPhysical = true;

	printf(
		"Debug: Charge #%d (%.0f, %.0f, %.0f)\n", chargeCount - 1,
		charges[chargeCount - 1].x, charges[chargeCount - 1].y,
		charges[chargeCount - 1].charge
	);
	printf("Charges %d\n", chargeCount);

	HANDLE_ERROR(
		hipMemcpy(dev_charges, charges, chargeCount * sizeof(Particle), hipMemcpyHostToDevice)
	);
	HANDLE_ERROR(
		hipMemcpyToSymbol(HIP_SYMBOL(dev_chargeCount), &chargeCount, sizeof(chargeCount))
	);
}


void onMouseEvent(int button, int state, int x, int y) {
	if (button == GLUT_MIDDLE_BUTTON && state == GLUT_DOWN) {
		chargeCount = 0;
		HANDLE_ERROR(
			hipMemcpyToSymbol(HIP_SYMBOL(dev_chargeCount), &chargeCount, sizeof(chargeCount))
		);
		uchar4* dev_screen;
		size_t size;

		HANDLE_ERROR(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
		HANDLE_ERROR(
			hipGraphicsResourceGetMappedPointer((void**)&dev_screen, &size, cuda_vbo_resource)
		);
		dev_clearFrame<<<blocks, threads >>>(dev_screen);
		HANDLE_ERROR(hipDeviceSynchronize());
		HANDLE_ERROR(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
		glutPostRedisplay();
		return;
	}
	
	if (button != GLUT_LEFT_BUTTON) return;

	// Drag, start dragging
	/*if (state == GLUT_DOWN && selectedChargeIndex != -1) {
		isDragging = true;
		printf(
			"Drag particle #%d with charge %.2f... ", 
			selectedChargeIndex, 
			charges[selectedChargeIndex].charge
		);
		charges[selectedChargeIndex].isPhysical = false;
		HANDLE_ERROR(hipMemcpy(
			dev_charges + selectedChargeIndex,
			charges + selectedChargeIndex,
			1 * sizeof(Particle),
			hipMemcpyHostToDevice
		));
		return;
	}*/
	
	if (state == GLUT_UP) {
		if (selectedChargeIndex != -1) { // Drop, stop dragging
			isDragging = false;
			charges[selectedChargeIndex].isPhysical = true;
			HANDLE_ERROR(hipMemcpy(
				dev_charges + selectedChargeIndex,
				charges + selectedChargeIndex,
				1 * sizeof(Particle),
				hipMemcpyHostToDevice
			));
			printf("Drop\n");
		} else {
			addCharge(x, WINDOW_HEIGHT - y);
		}
	}
}

void onMouseMove(int x, int y) {
	if (isDragging && selectedChargeIndex != -1) {
		if (x >= WINDOW_WIDTH) {
			charges[selectedChargeIndex].x = WINDOW_WIDTH - 1;
		} else if (x < 0) {
			charges[selectedChargeIndex].x = 0;
		} else {
			charges[selectedChargeIndex].x = x;
		}

		if (y >= WINDOW_HEIGHT) {
			charges[selectedChargeIndex].y = 0;
		} else if (y < 0) {
			charges[selectedChargeIndex].y = WINDOW_HEIGHT - 1;
		} else {
			charges[selectedChargeIndex].y = WINDOW_HEIGHT - y;
		}

		HANDLE_ERROR(
			hipMemcpy(
				dev_charges + selectedChargeIndex,
				charges + selectedChargeIndex,
				1 * sizeof(Particle),
				hipMemcpyHostToDevice
		));
	}
}

// Detect selected charge
void mouseTrack(int x, int y) {
	if (isDragging) return;

	HANDLE_ERROR(
		hipMemcpy(charges, dev_charges, chargeCount * sizeof(Particle), hipMemcpyDeviceToHost)
	);

	for (int i = 0; i < chargeCount; i++) {
		int dx = x - charges[i].x;
		int dy = (WINDOW_HEIGHT - y) - charges[i].y;

		if (charges[i].x >= 10 * WINDOW_WIDTH) return;
		if (charges[i].x < -10 * WINDOW_WIDTH) return;
		if (charges[i].y >= 10 * WINDOW_HEIGHT) return;
		if (charges[i].y < -10 * WINDOW_HEIGHT) return;

		if (dx * dx + dy * dy < DETECT_CHARGE_RANGE * DETECT_CHARGE_RANGE) {
			selectedChargeIndex = i;
			printf("#%d, x: %f, y: %f\n", selectedChargeIndex, charges[i].x, charges[i].y);
			return;
		}
	}

	selectedChargeIndex = -1;
}

void initCuda(int deviceId) {
	hipDeviceProp_t properties;
	HANDLE_ERROR(hipGetDeviceProperties(&properties, deviceId));

	threads.x = 32;
	// to avoid hipErrorLaunchOutOfResources error
	threads.y = properties.maxThreadsPerBlock / threads.x - 2;

	blocks.x = (WINDOW_WIDTH + threads.x - 1) / threads.x;
	blocks.y = (WINDOW_HEIGHT + threads.y - 1) / threads.y;

	hipMalloc(&dev_charges, sizeof(Particle) * MAX_CHARGE_COUNT);

	printf(
		"Debug: blocks(%d, %d), threads(%d, %d)\nCalculated Resolution: %d x %d\n",
		blocks.x, blocks.y, threads.x, threads.y, blocks.x * threads.x,
		blocks.y * threads.y
	);
}

void initGlut(int argc, char** argv) {
	// Initialize freeglut
	glutInit(&argc, argv);
	
	//glutInitDisplayMode(GLUT_RGBA);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	
	glutInitWindowSize(WINDOW_WIDTH, WINDOW_HEIGHT);
	int posX = glutGet(GLUT_SCREEN_WIDTH) / 2 - WINDOW_WIDTH / 2;
	int posY = glutGet(GLUT_SCREEN_HEIGHT) / 2 - WINDOW_HEIGHT / 2;
	glutInitWindowPosition(posX, posY);
	glutCreateWindow("Lab-4");

	glutIdleFunc(idle);
	glutDisplayFunc(draw);
	glutKeyboardFunc(onKeyEvent);
	glutMouseFunc(onMouseEvent);
	glutMotionFunc(onMouseMove);
	glutPassiveMotionFunc(mouseTrack);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0, (GLdouble)WINDOW_WIDTH, 0.0, (GLdouble)WINDOW_HEIGHT);

	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glEnable(GL_BLEND);

	glewInit();
}

int main(int argc, char** argv) {
	srand(time(0));
	initCuda(0);
	initGlut(argc, argv);

	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	glutMainLoop();

	deleteVBO(&vbo, cuda_vbo_resource);

	return 0;
}

static void cudaCheckError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

void createVBO(
	GLuint* vbo,
	struct hipGraphicsResource** vbo_res,
	unsigned int vbo_res_flags
) {
	unsigned int size = WINDOW_WIDTH * WINDOW_HEIGHT * sizeof(uchar4);

	glGenBuffers(1, vbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, *vbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, size, NULL, GL_DYNAMIC_DRAW);

	HANDLE_ERROR(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));
}

void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res) {
	HANDLE_ERROR(hipGraphicsUnregisterResource(cuda_vbo_resource));

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}