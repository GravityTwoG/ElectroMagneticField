#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <GL/glew.h>
#include <GL/freeglut.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>


const GLint WINDOW_WIDTH = 820;
const GLint WINDOW_HEIGHT = 640;
const int4 magneticField = { WINDOW_WIDTH / 3, 0, WINDOW_WIDTH, WINDOW_HEIGHT };

__device__ const float starB = 1e-9; // Tsl
__device__ const float starE = 1e-5; // V/m
__device__ const float C = 3e8;      // m/s
__device__ const float lambda = 3;// m
__device__ const int4 d_magneticField = { 
	WINDOW_WIDTH/3, // x-start
	-WINDOW_HEIGHT*10,              // y-start
	WINDOW_WIDTH*10,   // x-end
	WINDOW_HEIGHT*10   // y-end
};

const float TIME_SCALE = 0.1;
const float starV = 30000; // m/s
const float V_MIN = 0.5 * starV / C;
const float V_MAX = starV / C;

/* charge constants */
__constant__ const float K = 2e21;
__constant__ float MIN_DISTANCE = 1.0f; // not to divide by zero

const float MAX_CHARGE = 1.6e-19;
const float MIN_CHARGE = 0.3 * MAX_CHARGE;
const char MAX_CHARGE_COUNT = 10;

char chargeCount = 0;
__constant__ char dev_chargeCount;

struct Particle {
	float x;
	float y;
	float vx;
	float vy;
	float charge;
	float mass;
	bool isPhysical;
};

Particle charges[MAX_CHARGE_COUNT]; 
Particle* dev_charges;

/* OpenGL interoperability */
dim3 blocks, threads;
GLuint vbo;
hipGraphicsResource* cuda_vbo_resource;

static void cudaCheckError(hipError_t err, const char* file, int line);
#define HANDLE_ERROR( err ) (cudaCheckError( err, __FILE__, __LINE__ ))

void createVBO(GLuint* vbo, hipGraphicsResource** vbo_res, unsigned int vbo_res_flags);
void deleteVBO(GLuint* vbo, hipGraphicsResource* vbo_res);

void onKeyEvent(unsigned char key, int x, int y) {
	switch (key) {
	case 27:
		printf("Exit application\n");

		glutLeaveMainLoop();
		break;
	}
}

__device__ float length(const float2& q) {
	return sqrtf(q.x * q.x + q.y * q.y);
}

__device__ float length2(const float2& q) {
	return q.x * q.x + q.y * q.y;
}

__device__ bool isInMagneticField(float x, float y) {
	if (x < d_magneticField.x) return false;
	if (x > d_magneticField.z) return false;
	if (y < d_magneticField.y) return false;
	if (y > d_magneticField.w) return false;

	return true;
}

__device__ inline float4 dF(const Particle& p) {
	float B = starB * p.charge * lambda / (p.mass * C);
	float E = starE * p.charge * lambda / (p.mass * C * C);

	return {
		p.vx,
		p.vy,
		B * p.vy,
		E-B * p.vx
	};
}

// apply Columbus Law
__global__ void dev_applyMagneticField(uchar4* screen, Particle* dev_charges, float dt) {
	int charge_i = blockIdx.x * blockDim.x + threadIdx.x;
	if (charge_i >= MAX_CHARGE_COUNT) return;

	Particle& particle = dev_charges[charge_i];
	if (!particle.isPhysical) return;

	//float4 fi = dF(particle);
	//particle.x += dt * fi.x; // x + dx
	//particle.y += dt * fi.y; // y + dy
	//if (isInMagneticField(particle.x, particle.y)) {
	//	particle.vx += dt * fi.z;
	//	particle.vy += dt * fi.w;
	//}

	Particle p2 = particle;
	float4 d1 = dF(p2);

	p2.x = particle.x + dt * d1.x / 2;
	p2.y = particle.y + dt * d1.y / 2;
	p2.vx = particle.vx + dt * d1.z / 2;
	p2.vy = particle.vy + dt * d1.w / 2;
	float4 d2 = dF(p2);

	p2.x = particle.x + dt * d2.x / 2;
	p2.y = particle.y + dt * d2.y / 2;
	p2.vx = particle.vx + dt * d2.z / 2;
	p2.vy = particle.vy + dt * d2.w / 2;
	float4 d3 = dF(p2);
	
	p2.x = particle.x + dt * d3.x;
	p2.y = particle.y + dt * d3.y;
	p2.vx = particle.vx + dt * d3.z;
	p2.vy = particle.vy + dt * d3.w;
	float4 d4 = dF(p2);

	particle.x += dt / 6 * (d1.x + 2*d2.x + 2*d3.x + d4.x); // x + dx
	particle.y += dt / 6 * (d1.y + 2*d2.y + 2*d3.y + d4.y); // y + dy
	if (isInMagneticField(particle.x, particle.y)) {
		particle.vx += dt / 6 * (d1.z + 2 * d2.z + 2 * d3.z + d4.z);
		particle.vy += dt / 6 * (d1.w + 2 * d2.w + 2 * d3.w + d4.w);
	}

	if (particle.x >= 10 * WINDOW_WIDTH) particle.isPhysical = false;
	if (particle.x < -10 * WINDOW_WIDTH) particle.isPhysical = false;
	if (particle.y >= 10 * WINDOW_HEIGHT) particle.isPhysical = false;
	if (particle.y < -10 * WINDOW_HEIGHT) particle.isPhysical = false;

	if (
		particle.x < WINDOW_WIDTH && 
		particle.x >=  0 && 
		particle.y < WINDOW_HEIGHT && 
		particle.y >= 0
	) {
		uchar4& pixel = screen[(int)particle.x + (int)particle.y * WINDOW_WIDTH];
		//pixel.y = 150;
	}
}

__global__ void dev_clearFrame(uchar4* screen) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= WINDOW_WIDTH || y >= WINDOW_HEIGHT) return;

	uchar4& pixel = screen[x + y * WINDOW_WIDTH];
	pixel.x = 0;
	pixel.y = 0;
	pixel.z = 0;
	pixel.w = 255;
}

// Compute electric field
__global__ void dev_renderFrame(uchar4* screen, Particle* dev_charges) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= WINDOW_WIDTH || y >= WINDOW_HEIGHT) return;

	float2 force = { 0, 0 };

	float E = 0;
	float2 t_force;
	// iterate over all charges and compute resulted force vector
	for (char i = 0; i < dev_chargeCount; i++) {
		const Particle& particle = dev_charges[i];
		t_force.x = x - particle.x; // dx
		t_force.y = y - particle.y; // dy

		// x^2 + y^2
		float lengthSquared = length2(t_force) + MIN_DISTANCE;

		//e = q / (x^2 + y^2)^(3/2)
		float e = particle.charge / sqrtf(lengthSquared * lengthSquared * lengthSquared);
		E += e;
		t_force.x *= e;
		t_force.y *= e;

		force.x += t_force.x;
		force.y += t_force.y;
	}

	force.x *= K;
	force.y *= K;

	uchar4& pixel = screen[x + y * WINDOW_WIDTH];
	//pixel.x = pixel.z = 0;
	pixel.w = 255;

	float l = length(force); // 
	if (l < 70) return;

	float lScale = 2;
	int maxL = 255;
	if (E > 0.0) {
		pixel.x = l * lScale;
	} else {
		pixel.z = l * lScale;
	}
}

float elapsedTime = 0.0f;

void idle(void) {
	uchar4* dev_screen;
	size_t size;

	HANDLE_ERROR(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
	HANDLE_ERROR(
		hipGraphicsResourceGetMappedPointer((void**)&dev_screen, &size, cuda_vbo_resource)
	);

	// Kernel Time measure
	hipEvent_t startEvent, stopEvent;
	HANDLE_ERROR(hipEventCreate(&startEvent));
	HANDLE_ERROR(hipEventCreate(&stopEvent));
	HANDLE_ERROR(hipEventRecord(startEvent, 0));

	float elapsedTimeS = elapsedTime / 1000.0;
	//float elapsedTimeS = 1 / 1000.0;
	float dtau = elapsedTimeS * C / lambda;
	dev_applyMagneticField<<<1, MAX_CHARGE_COUNT>>>(
		dev_screen, dev_charges, 
		dtau * TIME_SCALE
	);
	dev_renderFrame<<<blocks, threads>>>(dev_screen, dev_charges);
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));

	// Kernel Time measure
	HANDLE_ERROR(hipEventRecord(stopEvent, 0));
	HANDLE_ERROR(hipEventSynchronize(stopEvent));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent));

	char fps[256];
	sprintf(fps, "%3.2f ms per frame (FPS: %3.1f)", elapsedTime,
		1000 / elapsedTime);
	glutSetWindowTitle(fps);

	glutPostRedisplay();
}

void draw(void) {
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	
	glDrawPixels(WINDOW_WIDTH, WINDOW_HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	glColor4f(0.4f, 0.4f, 1.0f, 0.2f);
	glRecti(
		magneticField.x, magneticField.y,
		magneticField.z, magneticField.w
	);

	glutSwapBuffers();
}

void clearScreen() {
	for (int i = 0; i < chargeCount; i++) {
		charges[i].isPhysical = false;
	}
	HANDLE_ERROR(
		hipMemcpy(dev_charges, charges, chargeCount * sizeof(Particle), hipMemcpyHostToDevice)
	);
	chargeCount = 0;
	HANDLE_ERROR(
		hipMemcpyToSymbol(HIP_SYMBOL(dev_chargeCount), &chargeCount, sizeof(chargeCount))
	);
	uchar4* dev_screen;
	size_t size;

	HANDLE_ERROR(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
	HANDLE_ERROR(
		hipGraphicsResourceGetMappedPointer((void**)&dev_screen, &size, cuda_vbo_resource)
	);
	dev_clearFrame<<<blocks, threads>>>(dev_screen);
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
	glutPostRedisplay();
}

void addCharge(int x, int y) {
	HANDLE_ERROR(
		hipMemcpy(charges, dev_charges, chargeCount * sizeof(Particle), hipMemcpyDeviceToHost)
	);
	
	if (chargeCount < MAX_CHARGE_COUNT) {
		chargeCount++;
	} else {
		// remove first charge
		for (int i = 0; i < MAX_CHARGE_COUNT - 1; ++i) {
			charges[i] = charges[i + 1];
		}
	}

	float scale = rand() / (float)RAND_MAX; /* [0, 1.0] */
	float newCharge = MIN_CHARGE + (float)scale * (MAX_CHARGE - MIN_CHARGE);      /* [min, max] */

	float scale2 = rand() / (float)RAND_MAX; /* [0, 1.0] */
	if (scale2 < 0.5) {
		newCharge = -newCharge;
	}
	
	float vScale = rand() / (float)RAND_MAX; /* [0, 1.0] */

	charges[chargeCount - 1].x = x;
	charges[chargeCount - 1].y = y;
	charges[chargeCount - 1].charge = newCharge;
	charges[chargeCount - 1].vx = V_MIN + vScale * (V_MAX - V_MIN);
	charges[chargeCount - 1].vy = 0.0f;
	//charges[chargeCount - 1].mass = fabs(newCharge / 10e10);
	charges[chargeCount - 1].mass = 9e-31;
	charges[chargeCount - 1].isPhysical = true;

	printf(
		"Debug: Charge #%d (%.0f, %.0f, %.0f)\n", chargeCount - 1,
		charges[chargeCount - 1].x, charges[chargeCount - 1].y,
		charges[chargeCount - 1].charge
	);
	printf("Charges %d\n", chargeCount);

	HANDLE_ERROR(
		hipMemcpy(dev_charges, charges, chargeCount * sizeof(Particle), hipMemcpyHostToDevice)
	);
	HANDLE_ERROR(
		hipMemcpyToSymbol(HIP_SYMBOL(dev_chargeCount), &chargeCount, sizeof(chargeCount))
	);
}

void addCharges(int x, int y) {
	clearScreen();
	float disp = 40;
	for (int i = 0; i < MAX_CHARGE_COUNT; i++) {
		float dx = rand() / (float)RAND_MAX * disp;
		float dy = rand() / (float)RAND_MAX * disp;
		
		addCharge(x + dx - disp/2, y + dy - disp / 2);
	}
}


void onMouseEvent(int button, int state, int x, int y) {
	if (button == GLUT_MIDDLE_BUTTON && state == GLUT_DOWN) {
		clearScreen();
		return;
	}
	
	if (button == GLUT_RIGHT_BUTTON && state == GLUT_UP) {
		addCharge(x, WINDOW_HEIGHT - y);
		return;
	}

	if (button == GLUT_LEFT_BUTTON && state == GLUT_UP) {
		addCharges(x, WINDOW_HEIGHT - y);
		return;
	}
}

void onMouseMove(int x, int y) {
	
}

// Detect selected charge
void mouseTrack(int x, int y) {
	
}

void initCuda(int deviceId) {
	hipDeviceProp_t properties;
	HANDLE_ERROR(hipGetDeviceProperties(&properties, deviceId));

	threads.x = 32;
	// to avoid hipErrorLaunchOutOfResources error
	threads.y = properties.maxThreadsPerBlock / threads.x - 2;

	blocks.x = (WINDOW_WIDTH + threads.x - 1) / threads.x;
	blocks.y = (WINDOW_HEIGHT + threads.y - 1) / threads.y;

	hipMalloc(&dev_charges, sizeof(Particle) * MAX_CHARGE_COUNT);

	printf(
		"Debug: blocks(%d, %d), threads(%d, %d)\nCalculated Resolution: %d x %d\n",
		blocks.x, blocks.y, threads.x, threads.y, blocks.x * threads.x,
		blocks.y * threads.y
	);
}

void initGlut(int argc, char** argv) {
	// Initialize freeglut
	glutInit(&argc, argv);
	
	//glutInitDisplayMode(GLUT_RGBA);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	
	glutInitWindowSize(WINDOW_WIDTH, WINDOW_HEIGHT);
	int posX = glutGet(GLUT_SCREEN_WIDTH) / 2 - WINDOW_WIDTH / 2;
	int posY = glutGet(GLUT_SCREEN_HEIGHT) / 2 - WINDOW_HEIGHT / 2;
	glutInitWindowPosition(posX, posY);
	glutCreateWindow("Lab-4");

	glutIdleFunc(idle);
	glutDisplayFunc(draw);
	glutKeyboardFunc(onKeyEvent);
	glutMouseFunc(onMouseEvent);
	glutMotionFunc(onMouseMove);
	glutPassiveMotionFunc(mouseTrack);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0, (GLdouble)WINDOW_WIDTH, 0.0, (GLdouble)WINDOW_HEIGHT);

	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glEnable(GL_BLEND);

	glewInit();
}

int main(int argc, char** argv) {
	srand(time(0));
	initCuda(0);
	initGlut(argc, argv);

	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	glutMainLoop();

	deleteVBO(&vbo, cuda_vbo_resource);

	return 0;
}

static void cudaCheckError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

void createVBO(
	GLuint* vbo,
	struct hipGraphicsResource** vbo_res,
	unsigned int vbo_res_flags
) {
	unsigned int size = WINDOW_WIDTH * WINDOW_HEIGHT * sizeof(uchar4);

	glGenBuffers(1, vbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, *vbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, size, NULL, GL_DYNAMIC_DRAW);

	HANDLE_ERROR(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));
}

void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res) {
	HANDLE_ERROR(hipGraphicsUnregisterResource(cuda_vbo_resource));

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}